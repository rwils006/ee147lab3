
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE

__global__ void kernel_Hist(unsigned int* array, unsigned int size, unsigned int* histo,unsigned int num_bins){
		
	__shared__ unsigned int histo_private[2048];

	if(threadIdx.x<num_bins)histo_private[threadIdx.x] = 0;
	__syncthreads();


	int i = blockIdx.x*blockDim.x+threadIdx.x;
	int stride = blockDim.x*gridDim.x; //total number of threads
	
	while(i<size){
		atomicAdd(&(histo_private[array[i]]),1);
		i+=stride;
	}

	__syncthreads();
	
	if(threadIdx.x<num_bins){
		atomicAdd(&(histo[threadIdx.x]),histo_private[threadIdx.x]);
	}


}
/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void histogram(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins) {

	dim3 block(num_bins*num_bins);
	dim3 grid((num_elements + block.x - 1)/block.x);

	
	kernel_Hist<<<grid,block>>>(input,num_elements,bins,num_bins);


}


